#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "./src/library.h"
using namespace std;

// Seam Carving cu C++ GPU

int WIDTH;
__device__ int d_WIDTH;

int xSobel[3][3] = {{1,0,-1},{2,0,-2},{1,0,-1}};
int ySobel[3][3] = {{1,2,1},{0,0,0},{-1,-2,-1}};
__constant__ int d_xSobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
__constant__ int d_ySobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
const int filterWidth = 3;


__device__ uint8_t calculateGrayValue(const uchar3& pixel) {
    return (uint8_t)(0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z);
}



/**
 * @param argc[1] name of the input file (.pmn)
 * @param argc[2] name of output file with no extension, created by using host & device
 * @param argc[3] horizontal of image you want to resize 
 * @param argc[4] - optional - default(32): blocksize.x
 * @param argc[5] - optional - default(32): blocksize.y
 */
void checkInput(int argc, char ** argv, int &width, int &height, uchar3 * &rgbPic, int &desiredWidth, dim3 &blockSize) {
    if (argc != 4 && argc != 6) {
        printf("The number of arguments is invalid\n");
        exit(EXIT_FAILURE);
    }

    // Read file
    readPnm(argv[1], width, height, rgbPic);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    WIDTH = width;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &width, sizeof(int)));
    // Check user's desired width
    desiredWidth = atoi(argv[3]);

    if (desiredWidth <= 0 || desiredWidth >= width) {
        printf("Your desired width must between 0 & current picture's width!\n");
        exit(EXIT_FAILURE);
    }

    // Block size
    if (argc == 6) {
        blockSize.x = atoi(argv[4]);
        blockSize.y = atoi(argv[5]);
    } 


    // Check GPU is working or not
    printDeviceInfo();
}


__global__ void convertRgb2GrayKernel(uchar3 * rgbPic, int width, int height, uint8_t * grayPic) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width) {
        int i = r * width + c;
        grayPic[i] = 0.299f*rgbPic[i].x + 0.587f*rgbPic[i].y + 0.114f*rgbPic[i].z;
    }
}
__global__ void calEnergy2(uint8_t *inPixels, int width, int height, int *energy) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    extern __shared__ uint8_t s_inPixels[];

    // Load data into shared memory
    int s_col = threadIdx.x - filterWidth / 2;
    int s_row = threadIdx.y - filterWidth / 2;

    for (int i = s_row; i < blockDim.y + filterWidth - 1; i += blockDim.y) {
        for (int j = s_col; j < blockDim.x + filterWidth - 1; j += blockDim.x) {
            int tmpRow = min(max(row + i, 0), height - 1);
            int tmpCol = min(max(col + j, 0), width - 1);
            s_inPixels[(i + filterWidth / 2) * (blockDim.x + filterWidth - 1) + j + filterWidth / 2] = inPixels[tmpRow * width + tmpCol];
        }
    }

    __syncthreads();

    if (col < width && row < height) {
        int x_kernel = 0, y_kernel = 0;
        for (int i = 0; i < filterWidth; ++i) {
            for (int j = 0; j < filterWidth; ++j) {
                uint8_t closest = s_inPixels[(threadIdx.y + i) * (blockDim.x + filterWidth - 1) + threadIdx.x + j];
                int filterIdx = i * filterWidth + j;
                x_kernel += closest * d_xSobel[filterIdx];
                y_kernel += closest * d_ySobel[filterIdx];
            }
        }
        energy[row * width + col] = abs(x_kernel) + abs(y_kernel);
    }
}


__global__ void calEnergy(uint8_t * inPixels, int width, int height, int * energy) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int s_width = blockDim.x + filterWidth - 1;
    int s_height = blockDim.y + filterWidth - 1;

    // Each block loads data from GMEM to SMEM
    extern __shared__ uint8_t s_inPixels[];

    int readRow = row - filterWidth / 2, readCol, tmpRow, tmpCol;
    int firstReadCol = col - filterWidth / 2;
    int virtualRow, virtualCol;

    for (virtualRow = threadIdx.y; virtualRow < s_height; readRow += blockDim.y, virtualRow += blockDim.y) {
        tmpRow = readRow;
        readRow = min(max(readRow, 0), height - 1);//0 <= readCol <= height-1
        
        readCol = firstReadCol;
        virtualCol = threadIdx.x;

        for (; virtualCol < s_width; readCol += blockDim.x, virtualCol += blockDim.x) {
            tmpCol = readCol;
            readCol = min(max(readCol, 0), width - 1);// 0 <= readCol <= width-1
            
            s_inPixels[virtualRow * s_width + virtualCol] = inPixels[readRow * d_WIDTH + readCol];
            readCol = tmpCol;
        }
        readRow = tmpRow;
    } 
    __syncthreads();


    // Each thread compute energy on SMEM
    int x_kernel = 0, y_kernel = 0;
    for (int i = 0; i < filterWidth; ++i) {
        for (int j = 0; j < filterWidth; ++j) {
            uint8_t closest = s_inPixels[(threadIdx.y + i) * s_width + threadIdx.x + j];
            int filterIdx = i * filterWidth + j;
            x_kernel += closest * d_xSobel[filterIdx];
            y_kernel += closest * d_ySobel[filterIdx];
        }
    }

    // Each thread writes result from SMEM to GMEM
    if (col < width && row < height)
        energy[row * d_WIDTH + col] = abs(x_kernel) + abs(y_kernel);
}








__global__ void energyToTheEndKernel(int * energy, int * minimalEnergy, int width, int height, int fromRow) {
    size_t halfBlock = blockDim.x / 2;//blockDim.x >> 1

    int col = blockIdx.x * halfBlock - halfBlock + threadIdx.x;

    if (fromRow == 0 && col >= 0 && col < width) {
        minimalEnergy[col] = energy[col];
    }
    __syncthreads();

    for (int stride = fromRow != 0 ? 0 : 1; stride < halfBlock && fromRow + stride < height; ++stride) {
        if (threadIdx.x < blockDim.x - (stride << 1)) {
            int curRow = fromRow + stride;
            int curCol = col + stride;

            if (curCol >= 0 && curCol < width) {
                int idx = curRow * d_WIDTH + curCol;
                int aboveIdx = (curRow - 1) * d_WIDTH + curCol;

                int min = minimalEnergy[aboveIdx];
                if (curCol > 0 && minimalEnergy[aboveIdx - 1] < min)
                    min = minimalEnergy[aboveIdx - 1];
                
                if (curCol < width - 1 && minimalEnergy[aboveIdx + 1] < min)
                    min = minimalEnergy[aboveIdx + 1];
                

                minimalEnergy[idx] = min + energy[idx];
            }
        }
        __syncthreads();
    }
}


__global__ void findSeamAndRemoveKernel(uchar3 *outPixels, uint8_t *grayPixels, int * energy, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    int r = idx / width;
    int c = idx % width;

    // Tính minimal energy trên mỗi hàng của ảnh bằng kỹ thuật dynamic programming
    if (r > 0) {
        int minPrevEnergy = energy[(r-1)*width+c];
        if (c > 0) {
            minPrevEnergy = min(minPrevEnergy, energy[(r-1)*width+c-1]);
        }
        if (c < width - 1) {
            minPrevEnergy = min(minPrevEnergy, energy[(r-1)*width+c+1]);
        }
        energy[idx] = grayPixels[idx] + minPrevEnergy;
    }


    // Tìm kiếm seam và loại bỏ chúng bằng cách sử dụng minimal energy đã tính được
    if (r < height - 1) {
        __syncthreads();
        int minCol = c;
        if (c > 0 && energy[(r+1)*width+c-1] < energy[r*width+minCol]) {
            minCol = c - 1;
        }
        if (c < width - 1 && energy[(r+1)*width+c+1] < energy[r*width+minCol]) {
            minCol = c + 1;
        }
        __syncthreads();
        if (minCol == c) {
            outPixels[idx] = outPixels[idx + width];
            grayPixels[idx] = grayPixels[idx + width];
            energy[idx] = energy[idx + width];
        }
    }

}

__global__ void carvingKernel1(int * leastSignificantPixel, uchar3 * outPixels, uint8_t *grayPixels, int * energy, int width) {
    int row = blockIdx.x;
    int baseIdx = row * d_WIDTH;
    for (int i = leastSignificantPixel[row]; i < width - 1; ++i) {
        outPixels[baseIdx + i] = outPixels[baseIdx + i + 1];
        grayPixels[baseIdx + i] = grayPixels[baseIdx + i + 1];
        energy[baseIdx + i] = energy[baseIdx + i + 1];
    }
}

__global__ void carvingKernel2(int * leastSignificantPixel, uchar3 * outPixels, uint8_t *grayPixels, int * energy, int width) {
    int row = blockIdx.x;
    int leastSignificant = leastSignificantPixel[row];

    // Update pixels only for threads whose index is within the range of leastSignificant to width - 1
    for (int i = leastSignificant + threadIdx.x; i < width - 1; i += blockDim.x) {
        int baseIdx = row * d_WIDTH + i;
        outPixels[baseIdx] = outPixels[baseIdx + 1];
        grayPixels[baseIdx] = grayPixels[baseIdx + 1];
        energy[baseIdx] = energy[baseIdx + 1];
    }
}

__global__ void carvingKernel(int * leastSignificantPixel, uchar3 * outPixels, uint8_t *grayPixels, int * energy, int width) {
    __shared__ uchar3 sharedOutPixels[32];
    __shared__ uint8_t sharedGrayPixels[32];
    __shared__ int sharedEnergy[32];

    int row = blockIdx.x;
    int baseIdx = row * d_WIDTH;
    int leastSignificant = leastSignificantPixel[row];


    for (int i = leastSignificant + threadIdx.x; i < width - 1; i += blockDim.x) {
        int idx = baseIdx + i;

        // Copy a row of data into shared memory
        sharedOutPixels[threadIdx.x] = outPixels[idx + 1];
        sharedGrayPixels[threadIdx.x] = grayPixels[idx + 1];
        sharedEnergy[threadIdx.x] = energy[idx + 1];

        __syncthreads();

        // Compute values for the current row using the shared data
        if (i < width - 1) {
            outPixels[idx] = sharedOutPixels[threadIdx.x];
            grayPixels[idx] = sharedGrayPixels[threadIdx.x];
            energy[idx] = sharedEnergy[threadIdx.x];
        }

        __syncthreads();
    }
}

__global__ void carvingKernel4(int *leastSignificantPixel, uchar3 *outPixels, uint8_t *grayPixels, int *energy, int width) {
    __shared__ uchar3 sharedOutPixels[32];
    __shared__ uint8_t sharedGrayPixels[32];
    __shared__ int sharedEnergy[32];

    int row = blockIdx.x;
    int leastSignificant = leastSignificantPixel[row];
    int baseIdx = row * d_WIDTH + leastSignificant;
    int idx = baseIdx + threadIdx.x;

    // Load data into shared memory
    sharedOutPixels[threadIdx.x] = outPixels[idx];
    sharedGrayPixels[threadIdx.x] = grayPixels[idx];
    sharedEnergy[threadIdx.x] = energy[idx];

    __syncthreads();

    // Update pixels in the current row
    for (int i = leastSignificant + threadIdx.x; i < width - 1; i += blockDim.x) {
        outPixels[baseIdx + i] = outPixels[baseIdx + i + 1];
        grayPixels[baseIdx + i] = grayPixels[baseIdx + i + 1];
        energy[baseIdx + i] = energy[baseIdx + i + 1];
    }

    __syncthreads();

    // Write updated data back to global memory
    outPixels[idx] = sharedOutPixels[threadIdx.x];
    grayPixels[idx] = sharedGrayPixels[threadIdx.x];
    energy[idx] = sharedEnergy[threadIdx.x];
}


__global__ void findSeamKernel2(int *minimalEnergy, int *leastSignificantPixel, int width, int height) {
    int col = threadIdx.x;
    int aboveIdx, leftIdx, rightIdx;
    int currEnergy, leftEnergy, rightEnergy;

    for (int row = height - 1; row >= 0; --row) {
        aboveIdx = (row - 1) * d_WIDTH + col;
        currEnergy = minimalEnergy[row * d_WIDTH + col];
        leftEnergy = col > 0 ? minimalEnergy[aboveIdx + (col - 1)] : INT_MAX;
        rightEnergy = col < width - 1 ? minimalEnergy[aboveIdx + (col + 1)] : INT_MAX;

        if (leftEnergy <= currEnergy && leftEnergy <= rightEnergy) {
            col -= 1;
        } else if (rightEnergy <= currEnergy && rightEnergy <= leftEnergy) {
            col += 1;
        }
        leastSignificantPixel[row] = col;
    }
}

__global__ void findSeamKernel(int * minimalEnergy, int *leastSignificantPixel, int width, int height) {
    int minCol = 0, r = height - 1;

    for (int c = 1; c < width; ++c)
        if (minimalEnergy[r * d_WIDTH + c] < minimalEnergy[r * d_WIDTH + minCol])
            minCol = c;
    
    for (; r >= 0; --r) {
        leastSignificantPixel[r] = minCol;
        if (r > 0) {
            int aboveIdx = (r - 1) * d_WIDTH + minCol;
            int min = minimalEnergy[aboveIdx], minColCpy = minCol;

            if (minColCpy > 0 && minimalEnergy[aboveIdx - 1] < min) {
                min = minimalEnergy[aboveIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}






void findSeam(int * minimalEnergy, int *leastSignificantPixel, int width, int height) {
    int minCol = 0, r = height - 1;

    for (int c = 1; c < width; ++c)
        if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
            minCol = c;
    
    for (; r >= 0; --r) {
        leastSignificantPixel[r] = minCol;
        if (r > 0) {
            int aboveIdx = (r - 1) * WIDTH + minCol;
            int min = minimalEnergy[aboveIdx], minColCpy = minCol;

            if (minColCpy > 0 && minimalEnergy[aboveIdx - 1] < min) {
                min = minimalEnergy[aboveIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}


// HOST

int getPixelEnergy(uint8_t * grayPixels, int row, int col, int width, int height) {
    int x_kernel = 0;
    int y_kernel = 0;

    for (int i = 0; i < 3; ++i) { // 3: filter width
        for (int j = 0; j < 3; ++j) {
            int r = min(max(0, row - 1 + i), height - 1); // 0 <= row - 1 + i < height
            int c = min(max(0, col - 1 + j), width - 1); // 0 <= col - 1 + j < width

            uint8_t pixelVal = grayPixels[r * WIDTH + c];//

            x_kernel += pixelVal * xSobel[i][j];// Convolution with x-Sobel
            y_kernel += pixelVal * ySobel[i][j];// Convolution with y-Sobel
        }
    }
    return abs(x_kernel) + abs(y_kernel);// Add matrix
}



void energyToTheEnd(int * energy, int * minimalEnergy, int width, int height) {
    for (int c = 0; c < width; ++c) {
        minimalEnergy[c] = energy[c];
    }
    for (int r = 1; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            int idx = r * WIDTH + c;
            int aboveIdx = (r - 1) * WIDTH + c;

            int min = minimalEnergy[aboveIdx];
            if (c > 0 && minimalEnergy[aboveIdx - 1] < min) {
                min = minimalEnergy[aboveIdx - 1];
            }
            if (c < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                min = minimalEnergy[aboveIdx + 1];
            }

            minimalEnergy[idx] = min + energy[idx];
        }
    }
}

void hostResizing(uchar3 * inPixels, int width, int height, int desiredWidth, uchar3 * outPixels) {
    GpuTimer timer;
    timer.Start();

    memcpy(outPixels, inPixels, width * height * sizeof(uchar3));

    // Allocating memory
    int * energy = (int *)malloc(width * height * sizeof(int));
    int * minimalEnergy = (int *)malloc(width * height * sizeof(int));
    
    // Get grayscale
    uint8_t * grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    convertRgb2Gray_host(inPixels, width, height, grayPixels);

    // Calculate all pixels energy
    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            energy[r * WIDTH + c] = getPixelEnergy(grayPixels, r, c, width, height);
        }
    }

    while (width > desiredWidth) {
        // Calculate energy to the end. (go from bottom to top)
        energyToTheEnd(energy, minimalEnergy, width, height);

        // find min index of last row
        int minCol = 0, r = height - 1, prevMinCol;
        for (int c = 1; c < width; ++c) {
            if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
                minCol = c;
        }

        // Find and remove seam from last to first row
        for (; r >= 0; --r) {
            // remove seam pixel on row r
            for (int i = minCol; i < width - 1; ++i) {
                outPixels[r * WIDTH + i] = outPixels[r * WIDTH + i + 1];
                grayPixels[r * WIDTH + i] = grayPixels[r * WIDTH + i + 1];
                energy[r * WIDTH + i] = energy[r * WIDTH + i + 1];
            }

            // Update energy
            if (r < height - 1) {
                int affectedCol = max(0, prevMinCol - 2);

                while (affectedCol <= prevMinCol + 2 && affectedCol < width - 1) {
                    energy[(r + 1) * WIDTH + affectedCol] = getPixelEnergy(grayPixels, r + 1, affectedCol, width - 1, height);
                    affectedCol += 1;
                }
            }

            // find to the top
            if (r > 0) {
                prevMinCol = minCol;

                int aboveIdx = (r - 1) * WIDTH + minCol;
                int min = minimalEnergy[aboveIdx], minColCpy = minCol;
                if (minColCpy > 0 && minimalEnergy[aboveIdx - 1] < min) {
                    min = minimalEnergy[aboveIdx - 1];
                    minCol = minColCpy - 1;
                }
                if (minColCpy < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                    minCol = minColCpy + 1;
                }
            }
        }

        int affectedCol;
        for (affectedCol=max(0, minCol - 2); affectedCol <= minCol + 2 && affectedCol < width - 1; ++affectedCol) {
            energy[affectedCol] = getPixelEnergy(grayPixels, 0, affectedCol, width - 1, height);
        }

        --width;
    }
    
    free(grayPixels);
    free(minimalEnergy);
    free(energy);

    timer.Stop();
    timer.printTime((char *)"host");
}

//device

void deviceResizing(uchar3 * inPixels, int width, int height, int desiredWidth, uchar3 * outPixels, dim3 blockSize) {
    GpuTimer timer;
    timer.Start();
    // allocate kernel memory
    uchar3 * d_inPixels;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    uint8_t * d_grayPixels;
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    int * d_energy;
    CHECK(hipMalloc(&d_energy, width * height * sizeof(int)));
    int * d_leastSignificantPixel;
    CHECK(hipMalloc(&d_leastSignificantPixel, height * sizeof(int)));
    int * d_minimalEnergy;
    CHECK(hipMalloc(&d_minimalEnergy, width * height * sizeof(int)));

    // allocate host memory
    int * energy = (int *)malloc(width * height * sizeof(int));
    int * leastSignificantPixel = (int *)malloc(height * sizeof(int));
    int * minimalEnergy = (int *)malloc(width * height * sizeof(int));

    // dynamically sized smem used to compute energy
    size_t smemSize = ((blockSize.x + 3 - 1) * (blockSize.y + 3 - 1)) * sizeof(uint8_t);
    
    // block size use to calculate minimal energy to the end
    int blockSizeDp = 256;
    int gridSizeDp = (((width - 1) / blockSizeDp + 1) << 1) + 1;
    int stripHeight = (blockSizeDp >> 1) + 1;

    // copy input to device
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

    // turn input image to grayscale
    dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
    convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    while (width > desiredWidth) {
        // update energy
        calEnergy<<<gridSize, blockSize, smemSize>>>(d_grayPixels, width, height, d_energy);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // compute min seam table
        for (int i = 0; i < height; i += (stripHeight >> 1)) {
            energyToTheEndKernel<<<gridSizeDp, blockSizeDp>>>(d_energy, d_minimalEnergy, width, height, i);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());
        }

        // find least significant pixel index of each row and store in d_leastSignificantPixel (SEQUENTIAL, in kernel or host)
        // CHECK(hipMemcpy(minimalEnergy, d_minimalEnergy, WIDTH * height * sizeof(int), hipMemcpyDeviceToHost));
        // findSeam(minimalEnergy, leastSignificantPixel, width, height);
        int numThreadsPerBlock = 256;
        int numBlocks = (width + numThreadsPerBlock - 1) / numThreadsPerBlock;
        findSeamKernel<<<numBlocks, numThreadsPerBlock>>>(d_minimalEnergy, d_leastSignificantPixel, width, height);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // carve    
        // CHECK(hipMemcpy(d_leastSignificantPixel, leastSignificantPixel, height * sizeof(int), hipMemcpyHostToDevice));
        carvingKernel<<<height, 1>>>(d_leastSignificantPixel, d_inPixels, d_grayPixels, d_energy, width);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        
        --width;
    }

    CHECK(hipMemcpy(outPixels, d_inPixels, WIDTH * height * sizeof(uchar3), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_energy));
    CHECK(hipFree(d_leastSignificantPixel));
    CHECK(hipFree(d_minimalEnergy));

    free(minimalEnergy);
    free(leastSignificantPixel);
    free(energy);

    timer.Stop();
    timer.printTime((char *)"device");   
}

void deviceResizing2(uchar3 * inPixels, int width, int height, int desiredWidth, uchar3 * outPixels, dim3 blockSize) {
    GpuTimer timer;
    timer.Start();
    // allocate kernel memory
    uchar3 * d_inPixels;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    uint8_t * d_grayPixels;
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    int * d_energy;
    CHECK(hipMalloc(&d_energy, width * height * sizeof(int)));
    // allocate host memory
    int * energy = (int *)malloc(width * height * sizeof(int));
    // copy input to device
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

    // turn input image to grayscale
    dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
    size_t smemSize = ((blockSize.x + 3 - 1) * (blockSize.y + 3 - 1)) * sizeof(uint8_t);
    convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    int numSeamsToRemove = abs(width - desiredWidth);
    calEnergy<<<gridSize, blockSize, smemSize>>>(d_grayPixels, width, height, d_energy);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());
    for (int i = 0; i < numSeamsToRemove; ++i) {
        findSeamAndRemoveKernel<<<gridSize,blockSize>>>(d_inPixels, d_grayPixels, d_energy, width, height);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        --width;
    }

    CHECK(hipMemcpy(outPixels, d_inPixels, WIDTH * height * sizeof(uchar3), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_energy));

    free(energy);

    timer.Stop();
    timer.printTime((char *)"device");   
}

int main(int argc, char ** argv) {   

    int width, height, desiredWidth;
    uchar3 * rgbPic;
    dim3 blockSize(32, 32);

    // Check user's input
    checkInput(argc, argv, width, height, rgbPic, desiredWidth, blockSize);

    // HOST
    uchar3 * out_host = (uchar3 *)malloc(width * height * sizeof(uchar3));
    hostResizing(rgbPic, width, height, desiredWidth, out_host);

    // DEVICE
    uchar3 * out_device = (uchar3 *)malloc(width * height * sizeof(uchar3));
    deviceResizing(rgbPic, width, height, desiredWidth, out_device, blockSize);

    // Compute error
    printError((char * )"Error between device result and host result: ", out_host, out_device, width, height);

    // Write 2 results to files
    writePnm(out_host, desiredWidth, height, width, concatStr(argv[2], "_host.pnm"));
    writePnm(out_device, desiredWidth, height, width, concatStr(argv[2], "_device.pnm"));

    // Free memories
    free(rgbPic);
    free(out_host);
    free(out_device);
}